#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


 #include "mySeluPlugin.h"
 #include <hip/hip_fp16.h>

// SELU constants
constexpr float alpha = 1.6732632423543772848170429916717f;
constexpr float scale = 1.0507009873554804934193349852946f;

__global__ void mySeluKernel(
    const int N,
    const float* input,
    float* output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
         output[i] = input[i] >= 0 ? scale * input[i] : scale * alpha * (exp(input[i]) - 1);
    }
 }

 int mySeluInference(
     const int n,
     float* input,
     float* output,
     hipStream_t stream) {
    // NCHW
    const int nThreads = 512;

    int nBlocks = (n + nThreads - 1) / nThreads;

    mySeluKernel<<<nBlocks, nThreads, 0, stream>>>(n, input, output);

     hipError_t err = hipGetLastError();
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
         return 1;
     }
     return 0;
 }

 int mySeluPlugin::enqueue(
     int batchSize,
     const void* const* inputs,
     void** outputs,
     void* workspace,
     hipStream_t stream) {
    return mySeluInference(batchSize * mBatchDim, (float*)inputs[0], (float*)outputs[0], stream);
 }
